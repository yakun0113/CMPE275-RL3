
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void reduce1(int *g_idata, int *g_odata) {
  extern __shared__ int sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  sdata[tid] = g_idata[i];
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s>0; s>>=1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main(void)
{
  int N = 1<<28;

   int *input, *output;
   hipMallocManaged(&input, N * sizeof(int));
   hipMallocManaged(&output, N * sizeof(int));

  for (int i = 0; i < N; i++) {
    input[i] = 2;
    output[i] = 0;
  }

  int blockSize = 128;
  int numBlocks = (N + blockSize - 1) / blockSize;
  int smemSize = blockSize * sizeof(int);

  reduce1<<<numBlocks, blockSize, smemSize>>>(input, output);

  hipDeviceSynchronize();

  int final_result = 0;
  for (int i = 0; i < numBlocks; i++) {
    final_result += output[i];
  }
  std::cout << "final result = " << final_result << "\n";

  // Free memory
  hipFree(input);
  hipFree(output);

  return 0;
}
